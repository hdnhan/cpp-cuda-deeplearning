#include <hipblas.h>
#include <chrono>
#include "common.h"

typedef double T;  // float vs double
const uint64_t N = 1 << 28;

T dotvv(const T *ha, const T *hb, const uint64_t &N) {
    return common::dotvv(ha, hb, N);
}

float dot(const float *ha, const float *hb, const uint64_t &N) {
    // Allocate memory on the device-side (GPU-side)
    float *da, *db;
    float res = 0;

    hipMalloc(&da, N * sizeof(float));
    hipMalloc(&db, N * sizeof(float));

    // Copy data from the host to the device (CPU -> GPU)
    hipMemcpy(da, ha, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform the dot product
    hipblasSdot(handle, N, da, 1, db, 1, &res);

    // Free the device memory
    hipFree(da);
    hipFree(db);
    hipblasDestroy(handle);
    return res;
}

double dot(const double *ha, const double *hb, const uint64_t &N) {
    // Allocate memory on the device-side (GPU-side)
    double *da, *db;
    double res = 0;

    hipMalloc(&da, N * sizeof(double));
    hipMalloc(&db, N * sizeof(double));

    // Copy data from the host to the device (CPU -> GPU)
    hipMemcpy(da, ha, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N * sizeof(double), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform the dot product
    hipblasDdot(handle, N, da, 1, db, 1, &res);

    // Free the device memory
    hipFree(da);
    hipFree(db);
    hipblasDestroy(handle);
    return res;
}

int main(int argc, char *argv[]) {
    // Vectors for holding the host-side (CPU-side) data
    T *ha = (T *)malloc(N * sizeof(T));
    T *hb = (T *)malloc(N * sizeof(T));
    T res0 = 0, res1 = 0;

    // Initialize random numbers in each array
    auto start = std::chrono::high_resolution_clock::now();
    common::init(ha, N);
    common::init(hb, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<T> diff = end - start;
    printf("Randomization: %0.6fs\n", diff.count());

    start = std::chrono::high_resolution_clock::now();
    res0 = dotvv(ha, hb, N);
    end = std::chrono::high_resolution_clock::now();
    diff = end - start;
    printf("CPU-side     : %0.6fs\n", diff.count());

    start = std::chrono::high_resolution_clock::now();
    res1 = dot(ha, hb, N);
    end = std::chrono::high_resolution_clock::now();
    diff = end - start;
    printf("hipblasSdot   : %0.6fs\n", diff.count());
    std::cout << "Difference: " << res0 - res1 << std::endl;

    // Free host memory
    free(ha);
    free(hb);
    return 0;
}